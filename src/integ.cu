#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include "functions.h"

__global__ void func_kernel1d(float * dy, float* a, float* base, float * params, int n)
{
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t offset = idx; 
//      printf("%d, %d, %d, %d\n", idx, idy, idz, offset);
        // ensure we are within bounds

        float x[3] = {a[0] + base[0] * (0.5f + idx)};
        if (idx< n) {
                dy[offset] = F0(x, params);
        }
}


__global__ void func_kernel2d(float * dy, float* a, float* base, float * params, int n)
{
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
        uint32_t offset = idx + idy * blockDim.x * gridDim.x;
//	printf("%d, %d, %d\n", idx, idy, offset);
        // ensure we are within bounds

        float x[2] = {a[0] + base[0] * (0.5f + idx), a[1] + base[1] * (0.5f + idy)};
        if (idx< n && idy<n) {
                dy[offset] = F1(x, params);
        }
}

__global__ void func_kernel3dF2(float * dy, float* a, float* base, float * params, int n)
{
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idz = blockIdx.z * blockDim.z + threadIdx.z;
	uint32_t offset = idx + (idy + (blockDim.x * idz * gridDim.x))  * blockDim.x * gridDim.x; 
//	printf("%d, %d, %d, %d\n", idx, idy, idz, offset);
	// ensure we are within bounds

	float x[3] = {a[0] + base[0] * (0.5f + idx), a[1] + base[1] * (0.5f + idy), a[2] + base[2] * (0.5f + idz)};
	if (idx< n) {
		dy[offset] = F2(x, params) ;
	}
}

__global__ void func_kernel3dF3(float * dy, float* a, float* base, float * params, int n)
{
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
        uint32_t idz = blockIdx.z * blockDim.z + threadIdx.z;
        uint32_t offset = idx + (idy + (blockDim.x * idz * gridDim.x))  * blockDim.x * gridDim.x;
//      printf("%d, %d, %d, %d\n", idx, idy, idz, offset);
        // ensure we are within bounds

        float x[3] = {a[0] + base[0] * (0.5f + idx), a[1] + base[1] * (0.5f + idy), a[2] + base[2] * (0.5f + idz)};
        if (idx< n) {
                dy[offset] = F3(x, params) ;
        }
}

__global__ void func_kernel3dF4(float * dy, float* a, float* base, float * params, int n)
{
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
        uint32_t idz = blockIdx.z * blockDim.z + threadIdx.z;
        uint32_t offset = idx + (idy + (blockDim.x * idz * gridDim.x))  * blockDim.x * gridDim.x;
//      printf("%d, %d, %d, %d\n", idx, idy, idz, offset);
        // ensure we are within bounds

        float x[3] = {a[0] + base[0] * (0.5f + idx), a[1] + base[1] * (0.5f + idy), a[2] + base[2] * (0.5f + idz)};
        if (idx< n) {
                dy[offset] = F4(x, params) ;
        }
}

__global__ void func_kernel3dF5(float * dy, float* a, float* base, float * params, int n)
{
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
        uint32_t idz = blockIdx.z * blockDim.z + threadIdx.z;
        uint32_t offset = idx + (idy + (blockDim.x * idz * gridDim.x))  * blockDim.x * gridDim.x;
//      printf("%d, %d, %d, %d\n", idx, idy, idz, offset);
        // ensure we are within bounds

        float x[3] = {a[0] + base[0] * (0.5f + idx), a[1] + base[1] * (0.5f + idy), a[2] + base[2] * (0.5f + idz)};
        if (idx< n) {
                dy[offset] = F5(x, params) ;
        }
}

__global__ void func_kernel3dF6(float * dy, float* a, float* base, float * params, int n)
{
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
        uint32_t idz = blockIdx.z * blockDim.z + threadIdx.z;
        uint32_t offset = idx + (idy + (blockDim.x * idz * gridDim.x))  * blockDim.x * gridDim.x;
//      printf("%d, %d, %d, %d\n", idx, idy, idz, offset);
//	printf("%d, %d, %d, %d, %d\n", idx, idy, idz, offset, n);
        // ensure we are within bounds

        float x[3] = {a[0] + base[0] * (0.5f + idx), a[1] + base[1] * (0.5f + idy), a[2] + base[2] * (0.5f + idz)};
        if (idx< n) {
                dy[offset] = F6(x, params) ;
        }
}

__global__ void func_kernel3dF9(float * dy, float* a, float* base, float * params, int n)
{
        uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;
        uint32_t idz = blockIdx.z * blockDim.z + threadIdx.z;
        uint32_t offset = idx + (idy + (blockDim.x * idz * gridDim.x))  * blockDim.x * gridDim.x;
//      printf("%d, %d, %d, %d\n", idx, idy, idz, offset);
        printf("%d, %d, %d, %d, %d\n", idx, idy, idz, offset, n);
        // ensure we are within bounds

        float x[3] = {a[0] + base[0] * (0.5f + idx), a[1] + base[1] * (0.5f + idy), a[2] + base[2] * (0.5f + idz)};
	printf("%0.10f, %0.10f, %0.10f\n", x[0], x[1],x[2]);
	printf("%0.10f, %0.10f, %0.10f\n", a[0], a[1],a[2]);
        if (idx< n) {
                dy[offset] = myfunc(x, params) ;
        }
}
void cudasafe( hipError_t error, char* message)
{
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

double Integrate(
    int functionCode, // Identifies the function (and dimensionality k)
    const float *a, // An array of k lower bounds
    const float *b, // An array of k upper bounds
    int n, // A target accuracy
    const float *params, // Parameters to function
    float *errorEstimate // Estimated error in integral
) 
{
	size_t freeMem = 0;
	size_t totalMem = 0;
	hipMemGetInfo(&freeMem, &totalMem);  
	printf("Memory avaliable: Free: %lu, Total: %lu\n",freeMem, totalMem);
	const int nsize = 10000000;
	const int sz = sizeof(float) * nsize;
	float *devicemem;
	hipMalloc((void **)&devicemem, sz);

	hipMemset(devicemem, 0, sz); // zeros all the bytes in devicemem
	int n0=n, n1=n, n2=n;	// By default use n points in each dimension
	int k;	
	switch(functionCode){
		case 0:	k=1;	break;
		case 1:	k=2;	break;
		case 2:	k=3;	break;
		case 3:	k=3;	break;
		case 4:	k=3;	break;
		case 5:	k=3;	break;
		case 6:	k=3;	break;
		case 9: k=3; 	break;
		default:
			fprintf(stderr, "Invalid function code.");
			exit(1);
	}
	
	// Collapse any dimensions we don't use
	if(k<3){
		n2=1;
	}
	if(k<2){
		n1=1;
	}
	// size, in bytes, of each vector
	size_t bytes = (n0*n1*n2)*sizeof(float);
		
	float *y = (float*)malloc(bytes);
	
	float base[3] = {(b[0] - a[0])/n, (b[1] - a[1])/n, (b[2] - a[2])/n};
	printf("base: %0.10f, %0.10f, %0.10f\n", base[0], base[1], base[2]);
	// allocate memory for each vector on GPU
	float * dy;
	float * dbase;
	float * da;
	float * dparams;
//	int  * dn;
	
	hipMalloc(&dy, bytes);
	hipMalloc(&dbase, sizeof(base));
//	hipMalloc((void**)&dn, sizeof(int));	
	hipMalloc(&da, k*sizeof(int));
	hipMalloc(&dparams, sizeof(params));

	hipMemcpy(dbase, base, sizeof(base), hipMemcpyHostToDevice);
	hipMemcpy(da, a, k*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dparams, params, sizeof(params), hipMemcpyHostToDevice);
//	hipMemcpy(dn,&n,sizeof(int), hipMemcpyHostToDevice);

	//kernel execute
	if (k==1) {

		printf("1D\n");
		// number of threads in each thread block
		int blockSize = 32;
		dim3 dimBlock(blockSize);

		// number of thread blocks in grid
		int gridSize = (int) ceil((float)n/blockSize);
		dim3 dimGrid(gridSize);

		func_kernel1d<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n);
	}
	else if (k==2) {
                // number of threads in each thread block
		printf("2D\n");
                int blockSize = 32;
                dim3 dimBlock(blockSize, blockSize);

                // number of thread blocks in grid
                int gridSize = (int) ceil((float)n/blockSize);
                dim3 dimGrid(gridSize, gridSize);

                func_kernel2d<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n);	

	}
	else { 
                // number of threads in each thread block
		printf("3D\n");
                int blockSize = 2;
                dim3 dimBlock(blockSize, blockSize, blockSize);

                // number of thread blocks in grid
                int gridSize = (int) ceil((float)n/blockSize);
                dim3 dimGrid(gridSize, gridSize, gridSize);
                if (functionCode==2)
                    func_kernel3dF2<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n);
                else if (functionCode==3)
                    func_kernel3dF3<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n);
                else if (functionCode==4)
                    func_kernel3dF4<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n);
                else if (functionCode==5)
                    func_kernel3dF5<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n);
                else if (functionCode==6)
                     func_kernel3dF6<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n); 
                else if (functionCode==9)
                     func_kernel3dF9<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n); 
                else {
                    fprintf(stderr, "Invalid function code.");
		}
	}

	
	//copy array back
	hipMemcpy(y, dy, bytes, hipMemcpyDeviceToHost);
	
	double sum = 0;
	for(uint32_t i=0; i<n0*n1*n2; i++) {
		sum += y[i];
	}
	for(int j=0; j<k; j++)
		sum *= base[j];
	printf("final result: %0.10f\n", sum);

	hipFree(dy);
	hipFree(da);
	hipFree(dbase);
	hipFree(dparams);
//	hipFree(dn);

	free(y);
	hipMemset(devicemem, 0, sz); // zeros all the bytes in devicemem
	return sum;
}

void testmyfunc(void) {
//        float a[3]={0,0,0};
        float a[3]={-1,-1,-1};
        float b[3]={2,2,2};
        float error;
        int n = 2;
        Integrate(9, a, b, n, NULL, &error);
}

void test0(void) {
        float a[1]={0};
        float b[1]={1};
        float error;
        int n = 256;
        Integrate(0, a, b, n, NULL, &error);
}

void test1(void) {
	float a[2]={0,0};
	float b[2]={1,1};
	float params[2]={0.5,0.5};
	float error;
	int n = 128; 
	Integrate(1, a, b, n, params, &error); 	
}

void test2(void) {
	float exact=9.48557252267795;	// Correct to about 6 digits
	float a[3]={-1,-1,-1};
	float b[3]={1,1,1};
	int n = 256;	
	float error;
	Integrate(2, a, b, n, NULL, &error); 	
}

void test3(void) {
	float exact=-7.18387139942142f;	// Correct to about 6 digits
	float a[3]={0,0,0};
	float b[3]={5,5,5};
	float params[1]={2};
	int n = 256;	
	float error;
	Integrate(3, a, b, n, params, &error); 	
}

void test4(void) {
        float exact=0.677779532970409f;	// Correct to about 8 digits
	float a[3]={-16,-16,-16};	// We're going to cheat, and assume -16=-infinity.
	float b[3]={1,1,1};
	// We're going to use the covariance matrix with ones on the diagonal, and
	// 0.5 off the diagonal.
	const float PI=3.1415926535897932384626433832795f;
	float params[10]={
		1.5, -0.5, -0.5,
		-0.5, 1.5, -0.5,
		-0.5, -0.5, 1.5,
		pow(2*PI,-3.0/2.0)*pow(0.5,-0.5) // This is the scale factor
	};
	int n = 64;
	float error;
        Integrate(4, a, b, n, params, &error);
}
void test5(void) {
	float exact=13.4249394627056;	// Correct to about 6 digits
	float a[3]={0,0,0};
	float b[3]={3,3,3};
        int n = 512;
        float error;
        Integrate(5, a, b, n, NULL, &error);
}
void test6(void) {

	float exact=   2.261955088165;
	float a[3]={-4,-4,-4};
	float b[3]={4,4,4};
	float params[2]={3,0.01};
        int n = 128;
        float error;
        Integrate(6, a, b, n, params, &error);
}

int main( int argc, char* argv[]) {
//    test0(); // works
//   test1();  // works
//    test3(); // works
	testmyfunc();
//    test4();
//    test2(); 
//    test5(); // works
//    test6();
}


