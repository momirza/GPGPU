#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "functions.h"
//CUDA kernel

//__device__ float F1(float x){
//	return sin(x); 
//}


__global__ void func_kernel(func* f, float * dy, float* a, float* base, int n, int func_type)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = idx + idy * blockDim.x * gridDim.x;

	float params[2]={0.5,0.5};

	// ensure we are within bounds
	float x[2] = {a[0] + base[0] * ((float)0.5 + (float)idx, a[0] + base[1] * ((float)0.5 + (float)idx)};
	if (idx<n && idy<n)
		dy[offset] = F1(x, params ) * base;
	__syncthreads();
}
void cudasafe( hipError_t error, char* message)
{
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

int main( int argc, char* argv[])
{
	int n = 4096; 
	// device input/output vectors
	
	// size, in bytes, of each vector
	size_t bytes = n*sizeof(float);
		
	float *y = (float*)malloc(bytes);
	
	// float a, b;
	int functionCode = atoi(argv[1]);
	// sscanf(argv[2], "%f", &a); 
	// sscanf(argv[3], "%f", &b);
 	
	float a[2]={0,0};
	float b[2]={1,1};

	float base[2] = {(b[0] - a[0]) / (float)n, (b[1] - a[1]) / (float)n};
	printf("%f\n", base);
	printf("%f\n", a);

	// allocate memory for each vector on GPU
	float * dy;
	hipMalloc(&dy, bytes);
	// allocate memory for params	
	// number of threads in each thread block
	int blockSize = 1024;
	dim3 dimBlock(blockSize, blockSize);

	// number of thread blocks in grid
	int gridSize = (int) ceil((float)n/blockSize);
	dim3 dimGrid(gridSize) = (gridSize, gridSize);
	
	//kernel execute
	func_kernel<<<dimGrid, dimBlock>>>(&dF0, dy, a, base, n, 0);
	
	//copy array back
	hipMemcpy(y, dy, bytes, hipMemcpyDeviceToHost);
	
	float sum = 0;

	for(int i=0; i<n; i++) {
		sum += y[i];
	}
	printf("final result: %f\n", sum);

	hipFree(dy);

	free(y);
	return 0;
}

