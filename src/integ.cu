#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "functions.h"
//CUDA kernel

//__device__ float F1(float x){
//	return sin(x); 
//}


__global__ void func_kernel(func* f, float * dy, float a, float base, int n, int func_type)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = idx + idy * blockDim.x * gridDim.x;

	float params[2]={0.5,0.5};

	// ensure we are within bounds
	float x[1] = {a + base * ((float)0.5 + (float)idx, a + base * ((float)0.5 + (float)idx)};
	if (idx<n && idy<n)
		dy[idx*idy + idx] = F1(x, params ) * base;
	__syncthreads();
}
void cudasafe( hipError_t error, char* message)
{
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

int main( int argc, char* argv[])
{
	int n = 4096; 
	// device input/output vectors
	
	// size, in bytes, of each vector
	size_t bytes = n*sizeof(float);
		
	float *y = (float*)malloc(bytes);
	
	float a, b;
	int functionCode = atoi(argv[1]);
	sscanf(argv[2], "%f", &a); 
	sscanf(argv[3], "%f", &b);
 
	float base = (b - a) / (float)n;
	printf("%f\n", base);
	printf("%f\n", a);

	// allocate memory for each vector on GPU
	float * dy;
	hipMalloc(&dy, bytes);
	// allocate memory for params	
	// number of threads in each thread block
	int blockSize = 1024;
	dim3 dimBlock(blockSize, blockSize);

	// number of thread blocks in grid
	int gridSize = (int) ceil((float)n/blockSize);
	dim3 dimGrid(gridSize) = (gridSize, gridSize);
	
	//kernel execute
	func_kernel<<<dimGrid, dimBlock>>>(&dF0, dy, a, base, n, 0);
	
	//copy array back
	hipMemcpy(y, dy, bytes, hipMemcpyDeviceToHost);
	
	float sum = 0;

	for(int i=0; i<n; i++) {
		sum += y[i];
	}
	printf("final result: %f\n", sum);

	hipFree(dy);

	free(y);
	return 0;
}

