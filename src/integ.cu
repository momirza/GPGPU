#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "functions.h"



__global__ void func_kernel(float * dy, float* a, float* base, float * params, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = idx + idy * blockDim.x * gridDim.x; 

	// ensure we are within bounds
	float x[2] = {a[0] + base[0] * (0.5f + (float)idx), a[1] + base[1] * (0.5f + (float)idy)};
	__syncthreads();
	if (idx< n && idy<n) {
		dy[offset] = F1(x, params) ;
		for (int j=0; j<2; j++) {
			dy[offset] *= base[j];
		}
	}
}
void cudasafe( hipError_t error, char* message)
{
   if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}

double Integrate(
    int functionCode, // Identifies the function (and dimensionality k)
    const float *a, // An array of k lower bounds
    const float *b, // An array of k upper bounds
    float n, // A target accuracy
    const float *params, // Parameters to function
    float *errorEstimate // Estimated error in integral
) 
{
	//int n = 32; 
	// device input/output vectors
	
	// size, in bytes, of each vector
	size_t bytes = (n*n)*sizeof(float);
		
	float *y = (float*)malloc(bytes);
	
 	

	float base[2] = {(b[0] - a[0]) / (float)n, (b[1] - a[1]) / (float)n};

	// allocate memory for each vector on GPU
	float * dy;
	float * dbase;
	float * da;
	float * dparams;
	hipMalloc(&dy, bytes);
	hipMalloc(&dbase, sizeof(base));
	hipMalloc(&da, sizeof(a));
	hipMalloc(&dparams, sizeof(params));
	// allocate memory for params	
	// number of threads in each thread block
	int blockSize = 32;
	dim3 dimBlock(blockSize, blockSize);

	// number of thread blocks in grid
	int gridSize = (int) ceil((float)n/blockSize);
	dim3 dimGrid(gridSize, gridSize);
	
	hipMemcpy(dbase, base, sizeof(base), hipMemcpyHostToDevice);
	hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(dparams, params, sizeof(params), hipMemcpyHostToDevice);
	//kernel execute
	func_kernel<<<dimGrid, dimBlock>>>(dy, da, dbase, dparams, n);
	
	//copy array back
	hipMemcpy(y, dy, bytes, hipMemcpyDeviceToHost);
	
	float sum = 0;

	for(int i=0; i<n*n; i++) {
		sum += y[i];
	}
	printf("final result: %f\n", sum);

	hipFree(dy);
	hipFree(da);
	hipFree(dbase);
	hipFree(dparams);

	free(y);

	return sum;
}

int main( int argc, char* argv[]) {
	float a[2]={0,0};
	float b[2]={1,1};
	float params[2]={0.5,0.5};
	float error;
	int n = atoi(argv[1]);
	Integrate(2, a, b, n, params, &error); 	
}
